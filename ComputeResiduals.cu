#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include "math.h"
#include "ComputeResiduals.h"
#include "GpuFunctions.h"
#include "FilesReading.h"
#include "ShellFunctions.h"
#include "ArrayUtils.h"
#include "BcMacros.h"
#include "BcMacros3D.h"
#include "GpuSum.h"

__host__ FLOAT_TYPE computeResidual2D(FLOAT_TYPE *f_d, FLOAT_TYPE *fTemp_d,
FLOAT_TYPE *temp9a_d, FLOAT_TYPE *temp9b_d, int m, int n) {
	dim3 bpg9((9 * m * n - 1) / THREADS + 1);

	cpu_sqsub(f_d, fTemp_d, temp9a_d, 9 * m * n);
	return sqrt(cpu_sum_h(temp9a_d, temp9b_d, 9 * m * n));
}
__host__ FLOAT_TYPE computeResidual3D(FLOAT_TYPE *f_d, FLOAT_TYPE *fTemp_d,
FLOAT_TYPE *temp19a_d, FLOAT_TYPE *temp19b_d, int m, int n, int h) {
	dim3 bpg19((int) (sqrt(19 * m * n * h - 1) / THREADS) + 1,
			(int) (sqrt(19 * m * n * h - 1) / THREADS) + 1);
	dim3 tpb(THREADS, THREADS);
	cpu_sqsub(f_d, fTemp_d, temp19a_d, 19 * m * n * h);
	return sqrt(cpu_sum_h(temp19a_d, temp19b_d, 19 * m * n * h));
}
__host__ FLOAT_TYPE computeNewResidual3D(FLOAT_TYPE *fn, FLOAT_TYPE *fnprev,
FLOAT_TYPE *f1, FLOAT_TYPE *temp19a_d, FLOAT_TYPE *temp19b_d, int m, int n,
		int h) {
	dim3 bpg19((int) (sqrt(19 * m * n * h - 1) / THREADS) + 1,
			(int) (sqrt(19 * m * n * h - 1) / THREADS) + 1);
	dim3 tpb(THREADS, THREADS);
	gpu_NewResidual<<<bpg19, tpb>>>(fn, fnprev, f1, temp19a_d, 19 * m * n * h);
	return cpu_max_h(temp19a_d, temp19b_d, 19 * m * n * h);
}


__host__ FLOAT_TYPE computeDragLift2D(int *bcMask_d, FLOAT_TYPE *dl_d,
FLOAT_TYPE *tempA_d, FLOAT_TYPE *tempB_d, int m, int n, int boundaryId) {
	dim3 bpg((int) sqrt((m * n - 1) / THREADS) + 1,
			(int) sqrt((m * n - 1) / THREADS) + 1, 1);
	cpu_cond_copy_mask2D(tempA_d, dl_d, bcMask_d, boundaryId,
			m * n);
	return cpu_sum_h(tempA_d, tempB_d, m * n);
}
__host__ FLOAT_TYPE computeDragLift3D(int *bcBoundId_d, FLOAT_TYPE *dl_d,
FLOAT_TYPE *tempA_d, FLOAT_TYPE *tempB_d, int m, int n, int h, int boundaryId) {
	dim3 bpg(sqrt((m * n * h - 1) / THREADS) + 1,
			sqrt((m * n * h - 1) / THREADS) + 1);
	cpu_cond_copy_mask3D(tempA_d, dl_d, bcBoundId_d,
			boundaryId, m * n);
	return cpu_sum_h(tempA_d, tempB_d, m * n);
}
__global__ void gpu_NewResidual(FLOAT_TYPE *fn, FLOAT_TYPE *fnprev,
FLOAT_TYPE *f1, FLOAT_TYPE *res, int size) {
	int blockId = blockIdx.x + blockIdx.y * gridDim.x;
	int ind = blockId * (blockDim.x * blockDim.y) + (threadIdx.y * blockDim.x)
			+ threadIdx.x;
	if (ind < size) {
		res[ind] = abs(abs(fn[ind] - fnprev[ind]) / f1[ind]);
	}
}
